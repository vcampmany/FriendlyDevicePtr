#include "hip/hip_runtime.h"
#include <vector>

#include "../DevicePtr.hpp"
#include "common.cuh"

typedef float buffer_t;

void runKernelExample1(std::vector<buffer_t> &v_a)
{
    // Create a new DevicePtr
    // the object takes care of allocating the necessary resources in the GPU
    // and to copy the data in *v_a* into it. The object will take care of
    // deallocating the GPU resources when it goes out of scope
    dptr::DevicePtr<buffer_t> a(v_a.data(), v_a.size());

    dim3 blockSize(256, 1, 1);
    dim3 gridSize(ceil(static_cast<float>(a.getCount())/blockSize.x),
                  1,
                  1);

    // Launch a dummy kernel on the data
    // The CUDA kernel receives the raw pointer to the GPU resource with obtained with *a.get()*
    // The kernel internally operates with the data as if the resource was directly allocated with hipMalloc
    dummyKernel<<<gridSize, blockSize>>>(a.get(), a.getCount());

    // Download the data to the CPU
    hipMemcpy(v_a.data(), a.get(), a.getBytes(), hipMemcpyDeviceToHost);

} // At this point (at the scope exit) the CUDA resource associated
  // with the DevicePtr object will automatically be freed

void runKernelExample2(std::vector<buffer_t> &v_a)
{
    // Create new DevicePtr
    // The object takes care of allocating the necessary resources in the GPU
    // The object will take care of deallocating the GPU resources when it goes out of scope
    dptr::DevicePtr<buffer_t> a(v_a.size());

    // Copy data into the DevicePtr resource
    // We have direct access with the raw GPU pointer,
    hipMemcpy(a.get(), v_a.data(), v_a.size() * sizeof(buffer_t), hipMemcpyHostToDevice);

    dim3 blockSize(256, 1, 1);
    dim3 gridSize(ceil(static_cast<float>(a.getCount())/blockSize.x),
                  1,
                  1);

    // Launch a dummy kernel on the data
    // The CUDA kernel receives the raw pointer to the GPU resource with obtained with *a.get()*
    // The kernel internally operates with the data as if the resource was directly allocated with hipMalloc
    dummyKernel<<<gridSize, blockSize>>>(a.get(), a.getCount());

    // Download the data to the CPU
    hipMemcpy(v_a.data(), a.get(), a.getBytes(), hipMemcpyDeviceToHost);

} // At this point (at the scope exit) the CUDA resource associated
  // with the DevicePtr object will automatically be freed


int main(int argc, char **argv)
{
    uint32_t count = 10;
    std::vector<buffer_t> v_a(count);
    fillVector(v_a);

    runKernelExample1(v_a);

    runKernelExample2(v_a);

    printVector(v_a);

    hipDeviceReset();
    return 1;
}
